
#include <hip/hip_runtime.h>
/**
 * Copyright 2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#define THREADS 1024
__global__ void CustomSquareKernel(float *input1, float *output, size_t size) {
  auto idx = blockIdx.x * THREADS + threadIdx.x;
  if (idx < size) {
    output[idx] = input1[idx] * input1[idx];
  }
}

extern "C" int CustomSquare(int nparam, void **params, int *ndims, int64_t **shapes, const char **dtypes, void *stream,
                            void *extra) {
  hipStream_t custream = static_cast<hipStream_t>(stream);
  if (nparam != 2) return 1;
  void *input1 = params[0];
  void *output = params[1];

  size_t size = 1;

  for (int i = 0; i < ndims[1]; i++) {
    size *= shapes[1][i];
  }
  int n = size / THREADS;
  for (int i = 0; i < nparam; i++) {
    if (strcmp(dtypes[i], "float32") != 0) {
      return 2;
    }
  }

  CustomSquareKernel<<<n + 1, THREADS, 0, custream>>>(static_cast<float *>(input1), static_cast<float *>(output), size);
  return 0;
}
